#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#define PIXELSIZE 4


double* ClusterCore;
unsigned int*  ClusterPixelCount;
double* ClusterColorSum;

__constant__ double DevClusterCore[4*32];

__global__ void PixelToCluster(unsigned char* Image, int ClusterCount, int Width, int Height, int* LastIter)
{
	int ElementsInStr=PIXELSIZE*Width;
	int Distance=0;
	int SelectedCluster=0;
	int a,b,c;  

	int i,j;

	for(int offset=4*(blockIdx.x*blockDim.x+threadIdx.x); offset<Height*ElementsInStr; offset+=4*(gridDim.x*blockDim.x))
	{
	   i=offset/ElementsInStr;
	   j=offset%ElementsInStr;

	   a=(DevClusterCore[0]-Image[i*ElementsInStr+j])*(DevClusterCore[0]-Image[i*ElementsInStr+j]);
	   b=(DevClusterCore[1]-Image[i*ElementsInStr+j+1])*(DevClusterCore[1]-Image[i*ElementsInStr+j+1]);
	   c=(DevClusterCore[2]-Image[i*ElementsInStr+j+2])*(DevClusterCore[2]-Image[i*ElementsInStr+j+2]);

	   Distance=a+b+c;
	   SelectedCluster=0;

	   for(int k=1; k<ClusterCount; k++)		
		{
			a=(DevClusterCore[k*PIXELSIZE]-Image[i*ElementsInStr+j])*(DevClusterCore[k*PIXELSIZE]-Image[i*ElementsInStr+j]);
			b=(DevClusterCore[k*PIXELSIZE+1]-Image[i*ElementsInStr+j+1])*(DevClusterCore[k*PIXELSIZE+1]-Image[i*ElementsInStr+j+1]);
			c=(DevClusterCore[k*PIXELSIZE+2]-Image[i*ElementsInStr+j+2])*(DevClusterCore[k*PIXELSIZE+2]-Image[i*ElementsInStr+j+2]);

			if((a+b+c)<Distance)
			{
		    	Distance=a+b+c;
				SelectedCluster=k;
			}
		}

	   if(Image[i*ElementsInStr+j+3]!=SelectedCluster) (*LastIter)=0;  
		Image[i*ElementsInStr+j+3]=SelectedCluster;


	}
}

__host__ void ClusterDustribution(unsigned char* Image, int ClusterCount, int Width, int Height, int* LastIter)
{
	int ElementsInStr=PIXELSIZE*Width;
	int Distance=0;
	int SelectedCluster=0;
	int a,b,c;  

	*LastIter=1;

	for(int i=0; i<Height; i++)
	{
		for(int j=0; j<ElementsInStr; j+=PIXELSIZE)
		{
			a=(ClusterCore[0]-Image[i*ElementsInStr+j])*(ClusterCore[0]-Image[i*ElementsInStr+j]);
			b=(ClusterCore[1]-Image[i*ElementsInStr+j+1])*(ClusterCore[1]-Image[i*ElementsInStr+j+1]);
			c=(ClusterCore[2]-Image[i*ElementsInStr+j+2])*(ClusterCore[2]-Image[i*ElementsInStr+j+2]);

            Distance=a+b+c;
			SelectedCluster=0;
			for(int k=1; k<ClusterCount; k++)		
			{
				a=(ClusterCore[k*PIXELSIZE]-Image[i*ElementsInStr+j])*(ClusterCore[k*PIXELSIZE]-Image[i*ElementsInStr+j]);
			    b=(ClusterCore[k*PIXELSIZE+1]-Image[i*ElementsInStr+j+1])*(ClusterCore[k*PIXELSIZE+1]-Image[i*ElementsInStr+j+1]);
				c=(ClusterCore[k*PIXELSIZE+2]-Image[i*ElementsInStr+j+2])*(ClusterCore[k*PIXELSIZE+2]-Image[i*ElementsInStr+j+2]);

				if((a+b+c)<Distance)
				{
					Distance=a+b+c;
					SelectedCluster=k;
				}
			}
			if(Image[i*ElementsInStr+j+3]!=SelectedCluster) *LastIter=0;  
			Image[i*ElementsInStr+j+3]=SelectedCluster;
		}	
	}

}

__host__ void ClusterOffset(unsigned char* Image, int ClusterCount, int Width, int Height) 
{
	for(int i=0; i<ClusterCount; i++)
		ClusterPixelCount[i]=0;

	for(int i=0; i<4*ClusterCount; i++)
		ClusterColorSum[i]=0;

	int ElementsInStr=PIXELSIZE*Width;
	int ClusterNum=0;

	for(int i=0; i<Height; i++)
	{
		for(int j=0; j<ElementsInStr; j+=PIXELSIZE)
		{
			ClusterNum=Image[i*ElementsInStr+j+3];
			ClusterPixelCount[ClusterNum]++;
			ClusterColorSum[ClusterNum*PIXELSIZE]+=Image[i*ElementsInStr+j];
			ClusterColorSum[ClusterNum*PIXELSIZE+1]+=Image[i*ElementsInStr+j+1];
			ClusterColorSum[ClusterNum*PIXELSIZE+2]+=Image[i*ElementsInStr+j+2];		    
		}
	}

	for(int i=0; i<ClusterCount; i++)
	{
		ClusterCore[i*PIXELSIZE]=ClusterColorSum[i*PIXELSIZE]/ClusterPixelCount[i];	
		ClusterCore[i*PIXELSIZE+1]=ClusterColorSum[i*PIXELSIZE+1]/ClusterPixelCount[i];	
		ClusterCore[i*PIXELSIZE+2]=ClusterColorSum[i*PIXELSIZE+2]/ClusterPixelCount[i];	
	}
}


int main()
{
   char InPath[256];
   char OutPath[256];

   scanf("%s", InPath);

   FILE* InPut = fopen(InPath, "rb");
    if (InPut == NULL)
    {
        fprintf(stderr, "Cannot open in.data");
        exit(0);
    }

	scanf("%s", OutPath);
	FILE* OutPut = fopen(OutPath, "wb");
    if (OutPut == NULL)
    {
        fprintf(stderr, "Cannot create out.data");
        exit(0);
    }

	int ClusterNumber;     

	scanf("%d", &ClusterNumber);

	int* Xcoords = (int*)malloc(ClusterNumber*sizeof(int)); 
	int* Ycoords = (int*)malloc(ClusterNumber*sizeof(int)); 

	for(int i=0; i<ClusterNumber; i++)
	{
		scanf("%d", &Ycoords[i]);
		scanf("%d", &Xcoords[i]);
	}


    ClusterCore = (double*)malloc(4*ClusterNumber*sizeof(double));
	ClusterPixelCount = (unsigned int*)malloc(ClusterNumber*sizeof(unsigned int));
	ClusterColorSum = (double*)malloc(4*ClusterNumber*sizeof(double));

	int Width;
	int Height;

	fread(&Width, sizeof(int), 1, InPut);
	fread(&Height, sizeof(int), 1, InPut);


	unsigned char* Image = (unsigned char*)malloc(4*Width*Height*sizeof(unsigned char));
	fread(Image, 4*Width*Height*sizeof(unsigned char), 1, InPut);


	unsigned char* Dev_Image;
	hipMalloc((void**)&Dev_Image, 4*Width*Height*sizeof(unsigned char));
	hipMemcpy(Dev_Image, Image, 4*Width*Height*sizeof(unsigned char), hipMemcpyHostToDevice);


	for(int i=0; i<ClusterNumber; i++)
	{
		ClusterCore[i*PIXELSIZE]=Image[4*Width*Xcoords[i]+PIXELSIZE*Ycoords[i]];
		
		ClusterCore[i*PIXELSIZE+1]=Image[4*Width*Xcoords[i]+PIXELSIZE*Ycoords[i]+1];

		ClusterCore[i*PIXELSIZE+2]=Image[4*Width*Xcoords[i]+PIXELSIZE*Ycoords[i]+2];

		ClusterCore[i*PIXELSIZE+3]=0;
	}
	
    int* NotLastIter;     
	int* HostNotLastIter = (int*)malloc(sizeof(int));
	*HostNotLastIter=1;


	hipMalloc((void**)&NotLastIter, sizeof(int));
	hipMemcpy(NotLastIter, HostNotLastIter, sizeof(int), hipMemcpyHostToDevice);
	
	hipMemcpyToSymbol(HIP_SYMBOL(DevClusterCore), ClusterCore, 4*ClusterNumber*sizeof(double));

      while(1)
	{
		PixelToCluster<<<128, 512>>>(Dev_Image, ClusterNumber, Width, Height, NotLastIter);
		hipMemcpy(HostNotLastIter, NotLastIter, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(Image, Dev_Image, 4*Width*Height*sizeof(unsigned char), hipMemcpyDeviceToHost);

		if((*HostNotLastIter)==1) break;

		ClusterOffset(Image, ClusterNumber, Width, Height);                   
		hipMemcpyToSymbol(HIP_SYMBOL(DevClusterCore), ClusterCore, 4*ClusterNumber*sizeof(double));
		*HostNotLastIter=1;
		hipMemcpy(NotLastIter, HostNotLastIter, sizeof(int), hipMemcpyHostToDevice);
	}
	

	fwrite(&Width, sizeof(int), 1 ,OutPut);
	fwrite(&Height, sizeof(int), 1, OutPut);
	fwrite(Image, 4*Width*Height*sizeof(unsigned char),1, OutPut);



	hipFree(NotLastIter);
	hipFree(Dev_Image);
	free(Image);
	free(ClusterColorSum);
	free(ClusterCore);
	free(ClusterPixelCount);	
    return 0;
}

